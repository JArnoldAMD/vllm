#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>

#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAContextLight.h>
#include <hip/hip_runtime.h>
#include <hipblaslt/hipblaslt.h>
#include <hipblaslt/hipblaslt-ext.hpp>

#define max_workspace_size 2 * 128 * 1024 * 1024

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)                                                                                                 \
    CHECK_CUDA(x);                                                                                                     \
    CHECK_CONTIGUOUS(x)

#ifndef CHECK_HIP_ERROR
#define CHECK_HIP_ERROR(error)                                                                                         \
    if (error != hipSuccess) {                                                                                         \
        fprintf(stderr, "Hip error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error, __FILE__, __LINE__);        \
        exit(EXIT_FAILURE);                                                                                            \
    }
#endif

#ifndef CHECK_HIPBLASLT_ERROR
#define CHECK_HIPBLASLT_ERROR(error)                                                                                   \
    if (error != HIPBLAS_STATUS_SUCCESS) {                                                                             \
        fprintf(                                                                                                       \
            stderr, "hipBLASLt error: '%s'(%d) at %s:%d\n", hipblasStatusToString(error), error, __FILE__, __LINE__);  \
        exit(EXIT_FAILURE);                                                                                            \
    }
#endif

torch::Tensor fp8_gemm(torch::Tensor& a, torch::Tensor& b, torch::Tensor& scaleA, torch::Tensor& scaleB,
    torch::Tensor& scaleD, int algo_idx)
{
    auto a_strides{a.strides()};
    auto b_strides{b.strides()};
    auto a_sizes{a.sizes()};
    auto b_sizes{b.sizes()};

    // CHECK_INPUT(a);
    // CHECK_INPUT(b);
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fnuz && b.dtype() == torch::kFloat8_e4m3fnuz,
        "The input tensors should be in fp8.");
    TORCH_CHECK(a.dim() == 2 && b.dim() == 2, "Input tensors must be 2-D.");
    TORCH_CHECK(a_sizes[1] == b_sizes[0], "a dim 1 must match b dim 0.");

    auto options{at::TensorOptions().dtype(torch::kFloat8_e4m3fnuz).device(at::kCUDA)};
    auto result{torch::empty({a_sizes[0], b_sizes[1]}, options)};

    constexpr bool transpose_result = true;
    bool transpose_a;
    bool transpose_b;
    if ((b_strides[0] == 1) && (b_strides[1] >= std::max<int64_t>(1, b_sizes[0]))) {
        transpose_b = false;
    } else if ((b_strides[1] == 1) && (b_strides[0] >= std::max<int64_t>(1, b_sizes[1]))) {
        transpose_b = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }
    if ((a_strides[0] == 1) && (a_strides[1] >= std::max<int64_t>(1, a_sizes[0]))) {
        transpose_a = false;
    } else if ((a_strides[1] == 1) && (a_strides[0] >= std::max<int64_t>(1, a_sizes[1]))) {
        transpose_a = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }

    if (transpose_result) {
        bool tmp = transpose_a;
        transpose_a = !transpose_b;
        transpose_b = !tmp;
        a_strides = b.strides();
        b_strides = a.strides();
        a_sizes = b.sizes();
        b_sizes = a.sizes();
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    int64_t m = a_sizes[transpose_result ? 1 : 0];
    int64_t k = a_sizes[transpose_result ? 0 : 1];
    int64_t n = b_sizes[transpose_result ? 0 : 1];

    void* d_a = static_cast<void*>((transpose_result ? b : a).data_ptr());
    void* d_b = static_cast<void*>((transpose_result ? a : b).data_ptr());
    void* d_d = static_cast<void*>(result.data_ptr());

    // void *d_scaleA, *d_scaleB, *d_workspace;
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleA, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleB, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_workspace, max_workspace_size));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleA, &(transpose_result ? scaleB : scaleA), sizeof(float), hipMemcpyHostToDevice));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleB, &(transpose_result ? scaleA : scaleB), sizeof(float), hipMemcpyHostToDevice));
    auto d_scaleA = transpose_result ? scaleB.data_ptr() : scaleA.data_ptr();
    auto d_scaleB = transpose_result ? scaleA.data_ptr() : scaleB.data_ptr();
    auto d_scaleD = scaleD.data_ptr();

    auto handle = at::cuda::getCurrentCUDABlasLtHandle();
    auto stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    hipblaslt_ext::GemmPreference gemmPref;
    gemmPref.setMaxWorkspaceBytes(0);
    hipblaslt_ext::Gemm gemm(handle, transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N, HIP_R_8F_E4M3_FNUZ, HIP_R_8F_E4M3_FNUZ, HIP_R_8F_E4M3_FNUZ,
        HIP_R_8F_E4M3_FNUZ, HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue epilogue{}; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a = d_a;
    inputs.b = d_b;
    inputs.c = d_d;
    inputs.d = d_d;
    inputs.alpha = &alpha;
    inputs.beta = &beta;
    inputs.scaleA = d_scaleA;
    inputs.scaleB = d_scaleB;
    inputs.scaleD = d_scaleD;
    gemm.setProblem(m, n, k, 1, epilogue, inputs);
    if (algo_idx == 0) {
        constexpr int request_solutions = 1024;
        std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
        heuristicResult.reserve(request_solutions);
        CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));
        static size_t solSize = 0;
        if (heuristicResult.size() != solSize) {
            std::cout << "fp8 sols: " << heuristicResult.size() << "\n";
            solSize = heuristicResult.size();
            for (auto& res : heuristicResult) {
                auto idx = hipblaslt_ext::getIndexFromAlgo(res.algo);
                std::cout << idx << "\n";
            }
        }
        TORCH_CHECK(!heuristicResult.empty(), "No valid solution found!");
        algo_idx = hipblaslt_ext::getIndexFromAlgo(heuristicResult[0].algo);
    }
    std::vector<int> algoIndex(1);
    algoIndex[0] = algo_idx;
    std::vector<hipblasLtMatmulHeuristicResult_t> tmpAlgo;
    TORCH_CUDABLAS_CHECK(hipblaslt_ext::getAlgosFromIndex(handle, algoIndex, tmpAlgo));

    CHECK_HIPBLASLT_ERROR(gemm.initialize(tmpAlgo[0].algo, nullptr));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));

    // hipFree(d_scaleA);
    // hipFree(d_scaleB);

    return result;
}

torch::Tensor bf8_gemm(torch::Tensor& a, torch::Tensor& b, torch::Tensor& scaleA, torch::Tensor& scaleB,
    torch::Tensor& scaleD, int algo_idx)
{
    auto a_strides{a.strides()};
    auto b_strides{b.strides()};
    auto a_sizes{a.sizes()};
    auto b_sizes{b.sizes()};

    // CHECK_INPUT(a);
    // CHECK_INPUT(b);
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fnuz && b.dtype() == torch::kFloat8_e4m3fnuz,
        "The input tensors should be in fp8.");
    TORCH_CHECK(a.dim() == 2 && b.dim() == 2, "Input tensors must be 2-D.");
    TORCH_CHECK(a_sizes[1] == b_sizes[0], "a dim 1 must match b dim 0.");

    auto options{at::TensorOptions().dtype(torch::kFloat8_e5m2fnuz).device(at::kCUDA)};
    auto result{torch::empty({a_sizes[0], b_sizes[1]}, options)};

    constexpr bool transpose_result = true;
    bool transpose_a;
    bool transpose_b;
    if ((b_strides[0] == 1) && (b_strides[1] >= std::max<int64_t>(1, b_sizes[0]))) {
        transpose_b = false;
    } else if ((b_strides[1] == 1) && (b_strides[0] >= std::max<int64_t>(1, b_sizes[1]))) {
        transpose_b = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }
    if ((a_strides[0] == 1) && (a_strides[1] >= std::max<int64_t>(1, a_sizes[0]))) {
        transpose_a = false;
    } else if ((a_strides[1] == 1) && (a_strides[0] >= std::max<int64_t>(1, a_sizes[1]))) {
        transpose_a = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }

    if (transpose_result) {
        bool tmp = transpose_a;
        transpose_a = !transpose_b;
        transpose_b = !tmp;
        a_strides = b.strides();
        b_strides = a.strides();
        a_sizes = b.sizes();
        b_sizes = a.sizes();
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    int64_t m = a_sizes[transpose_result ? 1 : 0];
    int64_t k = a_sizes[transpose_result ? 0 : 1];
    int64_t n = b_sizes[transpose_result ? 0 : 1];

    void* d_a = static_cast<void*>((transpose_result ? b : a).data_ptr());
    void* d_b = static_cast<void*>((transpose_result ? a : b).data_ptr());
    void* d_d = static_cast<void*>(result.data_ptr());

    // void *d_scaleA, *d_scaleB, *d_workspace;
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleA, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleB, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_workspace, max_workspace_size));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleA, &(transpose_result ? scaleB : scaleA), sizeof(float), hipMemcpyHostToDevice));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleB, &(transpose_result ? scaleA : scaleB), sizeof(float), hipMemcpyHostToDevice));
    auto d_scaleA = transpose_result ? scaleB.data_ptr() : scaleA.data_ptr();
    auto d_scaleB = transpose_result ? scaleA.data_ptr() : scaleB.data_ptr();
    auto d_scaleD = scaleD.data_ptr();

    auto handle = at::cuda::getCurrentCUDABlasLtHandle();
    auto stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    hipblaslt_ext::GemmPreference gemmPref;
    //gemmPref.setMaxWorkspaceBytes(max_workspace_size);
    hipblaslt_ext::Gemm gemm(handle, transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N, HIP_R_8F_E4M3_FNUZ, HIP_R_8F_E4M3_FNUZ, HIP_R_8F_E5M2_FNUZ,
        HIP_R_8F_E5M2_FNUZ, HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue epilogue{}; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a = d_a;
    inputs.b = d_b;
    inputs.c = d_d;
    inputs.d = d_d;
    inputs.alpha = &alpha;
    inputs.beta = &beta;
    inputs.scaleA = d_scaleA;
    inputs.scaleB = d_scaleB;
    inputs.scaleD = d_scaleD;
    gemm.setProblem(m, n, k, 1, epilogue, inputs);

    constexpr int request_solutions = 1024;
    std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
    heuristicResult.reserve(request_solutions);
    static size_t solSize = 0;
    CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));
    if (heuristicResult.size() != solSize) {
        solSize = heuristicResult.size();
        std::cout << "bf8 sols: " << heuristicResult.size() << "\n";
    }
    TORCH_CHECK(!heuristicResult.empty(), "No valid solution found!");
    TORCH_CHECK(algo_idx < heuristicResult.size());

    // In this sample, the workspace is already allocated with max_workspace_size
    // If not, calculate the needed workspace_size and allocate d_workspace here
    // uint64_t workspace_size = 0;
    // for(int i = 0; i < returnedAlgoCount; i++)
    //     workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);
    // CHECK_HIP_ERRORhipMalloc(&d_workspace, workspace_size));

    // Make sure to initialize every time when algo changes
    CHECK_HIPBLASLT_ERROR(gemm.initialize(heuristicResult[algo_idx].algo, nullptr));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));

    // hipFree(d_scaleA);
    // hipFree(d_scaleB);

    return result;
}

torch::Tensor fp8_gemm_16(
    torch::Tensor& a, torch::Tensor& b, torch::Tensor& scaleA, torch::Tensor& scaleB, int algo_idx)
{
    auto a_strides{a.strides()};
    auto b_strides{b.strides()};
    auto a_sizes{a.sizes()};
    auto b_sizes{b.sizes()};

    // CHECK_INPUT(a);
    // CHECK_INPUT(b);
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fnuz && b.dtype() == torch::kFloat8_e4m3fnuz,
        "The input tensors should be in fp8.");
    TORCH_CHECK(a.dim() == 2 && b.dim() == 2, "Input tensors must be 2-D.");
    TORCH_CHECK(a_sizes[1] == b_sizes[0], "a dim 1 must match b dim 0.");

    auto options{at::TensorOptions().dtype(torch::kFloat16).device(at::kCUDA)};
    auto result{torch::empty({a_sizes[0], b_sizes[1]}, options)};

    constexpr bool transpose_result = true;
    bool transpose_a;
    bool transpose_b;
    if ((b_strides[0] == 1) && (b_strides[1] >= std::max<int64_t>(1, b_sizes[0]))) {
        transpose_b = false;
    } else if ((b_strides[1] == 1) && (b_strides[0] >= std::max<int64_t>(1, b_sizes[1]))) {
        transpose_b = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }
    if ((a_strides[0] == 1) && (a_strides[1] >= std::max<int64_t>(1, a_sizes[0]))) {
        transpose_a = false;
    } else if ((a_strides[1] == 1) && (a_strides[0] >= std::max<int64_t>(1, a_sizes[1]))) {
        transpose_a = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }

    if (transpose_result) {
        bool tmp = transpose_a;
        transpose_a = !transpose_b;
        transpose_b = !tmp;
        a_strides = b.strides();
        b_strides = a.strides();
        a_sizes = b.sizes();
        b_sizes = a.sizes();
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    int64_t m = a_sizes[transpose_result ? 1 : 0];
    int64_t k = a_sizes[transpose_result ? 0 : 1];
    int64_t n = b_sizes[transpose_result ? 0 : 1];

    void* d_a = static_cast<void*>((transpose_result ? b : a).data_ptr());
    void* d_b = static_cast<void*>((transpose_result ? a : b).data_ptr());
    void* d_d = static_cast<void*>(result.data_ptr());

    // void *d_scaleA, *d_scaleB, *d_workspace;
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleA, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_scaleB, sizeof(float)));
    // CHECK_HIP_ERROR(hipMalloc(&d_workspace, max_workspace_size));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleA, &(transpose_result ? scaleB : scaleA), sizeof(float), hipMemcpyHostToDevice));
    // CHECK_HIP_ERROR(hipMemcpy(d_scaleB, &(transpose_result ? scaleA : scaleB), sizeof(float), hipMemcpyHostToDevice));
    auto d_scaleA = transpose_result ? scaleB.data_ptr() : scaleA.data_ptr();
    auto d_scaleB = transpose_result ? scaleA.data_ptr() : scaleB.data_ptr();

    auto handle = at::cuda::getCurrentCUDABlasLtHandle();
    auto stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    hipblaslt_ext::GemmPreference gemmPref;
    gemmPref.setMaxWorkspaceBytes(0);
    hipblaslt_ext::Gemm gemm(handle, transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N, HIP_R_8F_E4M3_FNUZ, HIP_R_8F_E4M3_FNUZ, HIP_R_16F, HIP_R_16F,
        HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue epilogue{}; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a = d_a;
    inputs.b = d_b;
    inputs.c = d_d;
    inputs.d = d_d;
    inputs.alpha = &alpha;
    inputs.beta = &beta;
    inputs.scaleA = d_scaleA;
    inputs.scaleB = d_scaleB;
    gemm.setProblem(m, n, k, 1, epilogue, inputs);
    if (algo_idx == 0) {
        constexpr int request_solutions = 1024;
        std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
        heuristicResult.reserve(request_solutions);
        CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));
        static size_t solSize = 0;
        if (heuristicResult.size() != solSize) {
            std::cout << "fp16 sols: " << heuristicResult.size() << "\n";
            solSize = heuristicResult.size();
            for (auto& res : heuristicResult) {
                auto idx = hipblaslt_ext::getIndexFromAlgo(res.algo);
                std::cout << idx << "\n";
            }
        }
        algo_idx = hipblaslt_ext::getIndexFromAlgo(heuristicResult[0].algo);
        TORCH_CHECK(!heuristicResult.empty(), "No valid solution found!");
    }
    std::vector<int> algoIndex(1);
    algoIndex[0] = algo_idx;
    std::vector<hipblasLtMatmulHeuristicResult_t> tmpAlgo;
    TORCH_CUDABLAS_CHECK(hipblaslt_ext::getAlgosFromIndex(handle, algoIndex, tmpAlgo));

    CHECK_HIPBLASLT_ERROR(gemm.initialize(tmpAlgo[0].algo, nullptr));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));

    // hipFree(d_scaleA);
    // hipFree(d_scaleB);

    return result;
}

torch::Tensor fp8_gemm_new(torch::Tensor& a, torch::Tensor& b, torch::Tensor& scaleA, torch::Tensor& scaleB,
    torch::Tensor& scaleC, int algo_idx)
{
    auto a_strides{a.strides()};
    auto b_strides{b.strides()};
    auto a_sizes{a.sizes()};
    auto b_sizes{b.sizes()};

    // CHECK_INPUT(a);
    // CHECK_INPUT(b);
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fnuz && b.dtype() == torch::kFloat8_e4m3fnuz,
        "The input tensors should be in fp8.");
    TORCH_CHECK(a.dim() == 2 && b.dim() == 2, "Input tensors must be 2-D.");
    TORCH_CHECK(a_sizes[1] == b_sizes[0], "a dim 1 must match b dim 0.");

    auto options{at::TensorOptions().dtype(torch::kFloat8_e4m3fnuz).device(at::kCUDA)};
    auto result{torch::empty({a_sizes[0], b_sizes[1]}, options)};

    constexpr bool transpose_result = true;
    bool transpose_a;
    bool transpose_b;
    if ((b_strides[0] == 1) && (b_strides[1] >= std::max<int64_t>(1, b_sizes[0]))) {
        transpose_b = false;
    } else if ((b_strides[1] == 1) && (b_strides[0] >= std::max<int64_t>(1, b_sizes[1]))) {
        transpose_b = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }
    if ((a_strides[0] == 1) && (a_strides[1] >= std::max<int64_t>(1, a_sizes[0]))) {
        transpose_a = false;
    } else if ((a_strides[1] == 1) && (a_strides[0] >= std::max<int64_t>(1, a_sizes[1]))) {
        transpose_a = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }

    if (transpose_result) {
        bool tmp = transpose_a;
        transpose_a = !transpose_b;
        transpose_b = !tmp;
        a_strides = b.strides();
        b_strides = a.strides();
        a_sizes = b.sizes();
        b_sizes = a.sizes();
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    int64_t m = a_sizes[transpose_result ? 1 : 0];
    int64_t k = a_sizes[transpose_result ? 0 : 1];
    int64_t n = b_sizes[transpose_result ? 0 : 1];

    void* d_a = static_cast<void*>((transpose_result ? b : a).data_ptr());
    void* d_b = static_cast<void*>((transpose_result ? a : b).data_ptr());
    void* d_d = static_cast<void*>(result.data_ptr());

    auto d_scaleA = transpose_result ? scaleB.data_ptr() : scaleA.data_ptr();
    auto d_scaleB = transpose_result ? scaleA.data_ptr() : scaleB.data_ptr();
    auto d_scaleC = scaleC.data_ptr();

    auto handle = at::cuda::getCurrentCUDABlasLtHandle();
    auto stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    //////////////////////////
    hipblasOperation_t opa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    hipblasLtMatmulDesc_t computeDesc;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescCreate(&computeDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(computeDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opa, sizeof(int32_t)));
    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(computeDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opb, sizeof(int32_t)));

    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &d_scaleA, sizeof(float**)));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &d_scaleB, sizeof(float**)));

    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scaleC, sizeof(float**)));

    int64_t mat1_ld = a_strides[(transpose_a == transpose_result) ? 1 : 0];
    int64_t mat2_ld = b_strides[(transpose_b == transpose_result) ? 1 : 0];
    int64_t result_ld = result.stride(transpose_result ? 0 : 1);
    hipblasLtMatrixLayout_t Adesc;
    if (!transpose_a) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3_FNUZ, m, k, mat1_ld));
    } else {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3_FNUZ, k, m, mat1_ld));
    }
    hipblasLtMatrixLayout_t Bdesc;
    if (!transpose_b) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3_FNUZ, k, n, mat2_ld));
    } else {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3_FNUZ, n, k, mat2_ld));
    }
    hipblasLtMatrixLayout_t Cdesc;
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_8F_E4M3_FNUZ, m, n, result_ld));

    hipblasLtMatmulPreference_t pref;
    hipblasLtMatmulPreferenceCreate(&pref);

    constexpr int requestedResults = 1024;
    hipblasLtMatmulHeuristicResult_t results[requestedResults];
    int numResults;
    if (algo_idx == 0) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
            handle, computeDesc, Adesc, Bdesc, Cdesc, Cdesc, pref, requestedResults, results, &numResults));

        TORCH_CHECK(numResults > 0, "Could not find matmul solution");
        static int last_results = 0;
        if (last_results != numResults) {
            last_results = numResults;
            for (int i = 0; i < numResults; ++i) {
                auto idx = hipblaslt_ext::getIndexFromAlgo(results[i].algo);
                std::cout << idx << "\n";
            }
            std::cout << "fp8_new sols: " << numResults << "\n";
        }
    } else {
        std::vector<int> algoIndex(1);
        algoIndex[0] = algo_idx;
        std::vector<hipblasLtMatmulHeuristicResult_t> tmpAlgo;
        TORCH_CUDABLAS_CHECK(hipblaslt_ext::getAlgosFromIndex(handle, algoIndex, tmpAlgo));
        results[0] = tmpAlgo[0];
    }

    auto hipblasStatus_t = hipblasLtMatmul(handle, computeDesc, &alpha, d_a, Adesc, d_b, Bdesc, &beta, d_d, Cdesc, d_d,
        Cdesc, &(results[0].algo), nullptr, 0, stream);

    TORCH_CHECK(
        hipblasStatus_t == HIPBLAS_STATUS_SUCCESS, "CUDA error: ", at::cuda::blas::_cublasGetErrorEnum(hipblasStatus_t));
    TORCH_CUDABLAS_CHECK(hipblasLtMatmulPreferenceDestroy(pref));
    TORCH_CUDABLAS_CHECK(hipblasLtMatmulDescDestroy(computeDesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Adesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Bdesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Cdesc));
    return result;
}

torch::Tensor fp8_gemm_16_new(torch::Tensor& a, torch::Tensor& b, torch::Tensor& scaleA, torch::Tensor& scaleB,
    int algo_idx)
{
    auto a_strides{a.strides()};
    auto b_strides{b.strides()};
    auto a_sizes{a.sizes()};
    auto b_sizes{b.sizes()};

    // CHECK_INPUT(a);
    // CHECK_INPUT(b);
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fnuz && b.dtype() == torch::kFloat8_e4m3fnuz,
        "The input tensors should be in fp8.");
    TORCH_CHECK(a.dim() == 2 && b.dim() == 2, "Input tensors must be 2-D.");
    TORCH_CHECK(a_sizes[1] == b_sizes[0], "a dim 1 must match b dim 0.");

    auto options{at::TensorOptions().dtype(torch::kFloat16).device(at::kCUDA)};
    auto result{torch::empty({a_sizes[0], b_sizes[1]}, options)};

    constexpr bool transpose_result = true;
    bool transpose_a;
    bool transpose_b;
    if ((b_strides[0] == 1) && (b_strides[1] >= std::max<int64_t>(1, b_sizes[0]))) {
        transpose_b = false;
    } else if ((b_strides[1] == 1) && (b_strides[0] >= std::max<int64_t>(1, b_sizes[1]))) {
        transpose_b = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }
    if ((a_strides[0] == 1) && (a_strides[1] >= std::max<int64_t>(1, a_sizes[0]))) {
        transpose_a = false;
    } else if ((a_strides[1] == 1) && (a_strides[0] >= std::max<int64_t>(1, a_sizes[1]))) {
        transpose_a = true;
    } else {
        assert(false && "unusual strides detected, may need to clone a contiguous tensor");
    }

    if (transpose_result) {
        bool tmp = transpose_a;
        transpose_a = !transpose_b;
        transpose_b = !tmp;
        a_strides = b.strides();
        b_strides = a.strides();
        a_sizes = b.sizes();
        b_sizes = a.sizes();
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    int64_t m = a_sizes[transpose_result ? 1 : 0];
    int64_t k = a_sizes[transpose_result ? 0 : 1];
    int64_t n = b_sizes[transpose_result ? 0 : 1];

    void* d_a = static_cast<void*>((transpose_result ? b : a).data_ptr());
    void* d_b = static_cast<void*>((transpose_result ? a : b).data_ptr());
    void* d_d = static_cast<void*>(result.data_ptr());

    auto d_scaleA = transpose_result ? scaleB.data_ptr() : scaleA.data_ptr();
    auto d_scaleB = transpose_result ? scaleA.data_ptr() : scaleB.data_ptr();

    auto handle = at::cuda::getCurrentCUDABlasLtHandle();
    auto stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    //////////////////////////
    hipblasOperation_t opa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    hipblasLtMatmulDesc_t computeDesc;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescCreate(&computeDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(computeDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opa, sizeof(int32_t)));
    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(computeDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opb, sizeof(int32_t)));

    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &d_scaleA, sizeof(float**)));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &d_scaleB, sizeof(float**)));

    int64_t mat1_ld = a_strides[(transpose_a == transpose_result) ? 1 : 0];
    int64_t mat2_ld = b_strides[(transpose_b == transpose_result) ? 1 : 0];
    int64_t result_ld = result.stride(transpose_result ? 0 : 1);
    hipblasLtMatrixLayout_t Adesc;
    if (!transpose_a) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3_FNUZ, m, k, mat1_ld));
    } else {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3_FNUZ, k, m, mat1_ld));
    }
    hipblasLtMatrixLayout_t Bdesc;
    if (!transpose_b) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3_FNUZ, k, n, mat2_ld));
    } else {
        TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3_FNUZ, n, k, mat2_ld));
    }
    hipblasLtMatrixLayout_t Cdesc;
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, result_ld));

    hipblasLtMatmulPreference_t pref;
    hipblasLtMatmulPreferenceCreate(&pref);

    constexpr int requestedResults = 1024;
    hipblasLtMatmulHeuristicResult_t results[requestedResults];
    int numResults;
    if (algo_idx == 0) {
        TORCH_CUDABLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(
            handle, computeDesc, Adesc, Bdesc, Cdesc, Cdesc, pref, requestedResults, results, &numResults));

        TORCH_CHECK(numResults > 0, "Could not find matmul solution");
        static int last_results = 0;
        if (last_results != numResults) {
            last_results = numResults;
            for (int i = 0; i < numResults; ++i) {
                auto idx = hipblaslt_ext::getIndexFromAlgo(results[i].algo);
                std::cout << idx << "\n";
            }
            std::cout << "fp8_new sols: " << numResults << "\n";
        }
    } else {
        std::vector<int> algoIndex(1);
        algoIndex[0] = algo_idx;
        std::vector<hipblasLtMatmulHeuristicResult_t> tmpAlgo;
        TORCH_CUDABLAS_CHECK(hipblaslt_ext::getAlgosFromIndex(handle, algoIndex, tmpAlgo));
        results[0] = tmpAlgo[0];
    }

    auto hipblasStatus_t = hipblasLtMatmul(handle, computeDesc, &alpha, d_a, Adesc, d_b, Bdesc, &beta, d_d, Cdesc, d_d,
        Cdesc, &(results[0].algo), nullptr, 0, stream);

    TORCH_CHECK(
        hipblasStatus_t == HIPBLAS_STATUS_SUCCESS, "CUDA error: ", at::cuda::blas::_cublasGetErrorEnum(hipblasStatus_t));
    TORCH_CUDABLAS_CHECK(hipblasLtMatmulPreferenceDestroy(pref));
    TORCH_CUDABLAS_CHECK(hipblasLtMatmulDescDestroy(computeDesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Adesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Bdesc));
    TORCH_CUDABLAS_CHECK(hipblasLtMatrixLayoutDestroy(Cdesc));
    return result;
}